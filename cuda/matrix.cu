#include "hip/hip_runtime.h"
void initMat(){
	srand(time(NULL));
}

void matrixAllocate(matrix* mat, int m, int n){
	
	mat->m = m;
	mat->n = n;
	hipMalloc(&mat->array, sizeof(float)*m*n);
	
}

int matrixCopy(matrix* dst, matrix* src){
	
	if ((dst->n != src->n) || (dst->m != src->m)){
		printf("(%d, %d) != (%d, %d)\n",dst->m,dst->n,src->m,src->n);
		printf("matrixCopy wrong dimension\n");
		return -1;
	}
	int m = dst->m; int n = dst->n;
	
	hipMemcpy(dst->array, src->array, m*n * sizeof(float), hipMemcpyDeviceToDevice);
	
	return 0;
}

void matrixRandFill(matrix* mat){ //Do not call if anything has been initilized ther budrick
	
	int m = mat->m; int n = mat->n;
	float* temp = (float*)malloc(sizeof(float)*m*n);
 	for(int i = 0; i < m; i++){
		for(int j = 0; j < n; j++){
			temp[i*n + j] = gaussian();
		}
	}
	hipMemcpy(mat->array, temp, sizeof(float)*m*n,hipMemcpyHostToDevice);
	free(temp);
	
	return;
}

void matrixScalar(matrix* mat, float scalar){
	
	int m = mat->m; int n = mat->n;
	dim3 threadsPerBlock(32,32);
	dim3 numBlocks(ceil(float(n) / 32.0f), ceil(float(m) / 32.0f));
	matscalar<<<numBlocks, threadsPerBlock>>>(mat->array, scalar, m, n);
	
}
__global__ void matscalar(float* A, float scalar, int m, int n){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if((x < n) && (y < m)){
		A[y*n + x] *= scalar;
	}
}

int matrixTranspose(matrix* mat){
	int m = mat->m; int n = mat->n;
	float* newArray = NULL;
	hipMalloc(&newArray, sizeof(float) * m*n);
	dim3 threadsPerBlock(32,32);
	dim3 numBlocks(ceil(float(n)/32.0f), ceil(float(m)/32.0f));

	mattrans<<<numBlocks, threadsPerBlock>>>(newArray, mat->array, m,n);
	mat->n = mat->m;
	mat->m = n;
	hipFree(mat->array);
	mat->array = newArray;
	
	return 0;
}
__global__ void mattrans(float* dst, float* src, int m, int n){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if((x < n) && (y < m)){
		dst[x*m + y] = src[y*n + x];
	}
}

int matrixSigmoid(matrix* A){
	
	if (A->n != 1){
		perror("Cannot do sigmoid");
		return -1;
	};
	dim3 threadsPerBlock(32,32);
	dim3 numBlocks(ceil(float(A->m)/32.0f), ceil(float(A->n)/32.0f));
	
	matsig<<<numBlocks, threadsPerBlock>>>(A->array, A->m,A->n);
	
	return 0;
}
__global__ void matsig(float* dst, int m, int n){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if((x < n) && (y < m)){
		dst[y*n + x] = 1.0f/(1.0f + exp(-dst[y*n + x]));
	}
}

int matrixSigmoidPrime(matrix* A){
	
	if (A->n != 1){
		perror("Cannot do sigmoid prime");
		return -1;
	};
	dim3 threadsPerBlock(32,32);
	dim3 numBlocks(ceil(float(A->m)/32.0f), ceil(float(A->n)/32.0f));

	matsigp<<<numBlocks, threadsPerBlock>>>(A->array, A->m,A->n);
	return 0;
}
__global__ void matsigp(float* dst, int m, int n){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if((x < n) && (y < m)){
		float temp = dst[y*n + x];
		dst[y*n + x] = 1.0f/(2.0f + exp(-temp) + exp(temp));
	}
}

int matrixMultTransFirst(matrix* A, matrix* B, matrix* out){
	if ((A->m != B->m)||(A->n != out->m)||(B->n != out->n)){
		printf("(%d, %d)T x (%d, %d) != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}
	int am = A->n;
       	int an = A->m;
       	int bn = B->n; 
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	
	matmulttrans1<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	
	return 0;
}

__global__ void matmulttrans1(float* A, float* B, float* C, int m, int q, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float temp = 0;
	if((x < n) && (y < m)){
		for(int i = 0; i < q; i++){
			temp += A[i*m + y] * B[i*n + x];	
		}
	C[y*n + x] += temp;
	}
}

int matrixMult(matrix* A, matrix* B, matrix* out){
	
	if ((A->n != B->m)||(A->m != out->m)||(B->n != out->n)){
		printf("(%d, %d) x (%d, %d) != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}
	
	int am = A->m;
       	int an = A->n;
       	int bn = B->n; 
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	
	matmult<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	
	return 0;
}

__global__ void matmult(float* A, float* B, float* C, int m, int q, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float temp = 0;
	if((x < n) && (y < m)){
		for(int i = 0; i < q; i++){
			temp += A[y*q + i] * B[i*n + x];	
		}
	C[y*n + x] += temp;
	
	}
}

int matrixMultTransSecond(matrix* A, matrix* B, matrix* out){
	if ((A->n != B->n)||(A->m != out->m)||(B->m != out->n)){
		printf("(%d, %d) x (%d, %d)T != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}
	int am = A->m;
       	int an = A->n;
       	int bn = B->m; 
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	matmulttrans2<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	return 0;
}

__global__ void matmulttrans2(float* A, float* B, float* C, int m, int q, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float temp = 0;
	if((x < n) && (y < m)){
		for(int i = 0; i < q; i++){
			temp += A[y*q + i] * B[x*q + i];	
		}
	C[y*n + x] += temp;
	}
}

int matrixMultTransFirstAndDelete(matrix* A, matrix* B, matrix* out){
	if ((A->m != B->m)||(A->n != out->m)||(B->n != out->n)){
		printf("(%d, %d)T x (%d, %d) != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}
	int am = A->n;
       	int an = A->m;
       	int bn = B->n; 

	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	
	matmulttrans1D<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	
	return 0;
}

__global__ void matmulttrans1D(float* A, float* B, float* C, int m, int q, int n){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float temp = 0;
	if((x < n) && (y < m)){
		for(int i = 0; i < q; i++){
			temp += A[i*m + y] * B[i*n + x];	
		}
	C[y*n + x] = temp;
	}
}

int matrixMultAndDelete(matrix* A, matrix* B, matrix* out){
	
	if ((A->n != B->m)||(A->m != out->m)||(B->n != out->n)){
		printf("(%d, %d) x (%d, %d) != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}
	
	int am = A->m;
       	int an = A->n;
       	int bn = B->n; 
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	
	matmultD<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	
	return 0;
}

__global__ void matmultD(float* A, float* B, float* C, int m, int q, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float temp = 0;
	if((x < n) && (y < m)){
		for(int i = 0; i < q; i++){
			temp += A[y*q + i] * B[i*n + x];	
		}
	C[y*n + x] = temp;
	
	}
}

int matrixMultTransSecondAndDelete(matrix* A, matrix* B, matrix* out){
	if ((A->n != B->n)||(A->m != out->m)||(B->m != out->n)){
		printf("(%d, %d) x (%d, %d)T != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}

	int am = A->m;
       	int an = A->n;
       	int bn = B->m; 
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	
	matmulttrans2D<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	
	return 0;
}

__global__ void matmulttrans2D(float* A, float* B, float* C, int m, int q, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float temp = 0;
	if((x < n) && (y < m)){
		for(int i = 0; i < q; i++){
			temp += A[y*q + i] * B[x*q + i];	
		}
	C[y*n + x] = temp;
	}
}


int matrixAdd(matrix* dst, matrix* src){ //in place,a s opposed to matrix multiply???
	
	if ((dst->n != src->n) || (dst->m != src->m)){
		perror("matrixAdd wrong dimension");
		return -1;
	}
	int m = dst->m; int n = dst->n;
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(n) / 32.0f), ceil(float(m) / 32.0f));

	matadd<<<numBlocks, threadsPerBlock>>>(dst->array,src->array,m,n);
	
	return 0;
}

__global__ void matadd(float* A, float* B, int m, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if((x < n) && (y < m)){
		A[y*n + x] += B[y*n + x];	
	}
}

int matrixHamProd(matrix* dst, matrix* src){ //in place,a s opposed to matrix multiply???
	
	if ((dst->n != src->n) || (dst->m != src->m)){
		perror("matrixAdd wrong dimension");
		return -1;
	}
	int m = dst->m; int n = dst->n;
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(n) / 32.0f), ceil(float(m) / 32.0f));

	matham<<<numBlocks, threadsPerBlock>>>(dst->array,src->array,m,n);
	
	return 0;
}

__global__ void matham(float* A, float* B, int m, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if((x < n) && (y < m)){
		A[y*n + x] *= B[y*n +x];	
	}
}

void matrixPrint(matrix* mat) {
	hipDeviceSynchronize();	
	int m = mat->m; int n = mat->n;
	float* temp = (float*)malloc(sizeof(float)*m*n);
	
	hipMemcpy(temp, mat->array, sizeof(float)*m*n,hipMemcpyDeviceToHost);
	printf("m: %d n: %d \n", m,n);
    	for (int i = 0; i < m; i++) {
        	for (int j = 0; j < n; j++) {
        	  	  printf(" %lf ", temp[i*n + j]);
        	}
        	printf("\n");
    	}
	free(temp);
	
}

int matrixNonZeros(matrix* mat) {
	hipDeviceSynchronize();	
	int m = mat->m; int n = mat->n;
	float* temp = (float*)malloc(sizeof(float)*m*n);
	
	hipMemcpy(temp, mat->array, sizeof(float)*m*n,hipMemcpyDeviceToHost);
	int nonzeros = 0;
    	for (int i = 0; i < m; i++) {
        	for (int j = 0; j < n; j++) {
        		if(temp[i*n + j]){
				nonzeros++;
			}	
		}
    	}
	free(temp);
	printf("%d non zero \n", nonzeros);
	return nonzeros;
}
void matrixFree(matrix* mat) {
	
    hipFree(mat->array); // Free the array of rows
	
}


