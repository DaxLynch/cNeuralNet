
#include <hip/hip_runtime.h>
double uniformlyRandD(){ //Returns uniformly random on the interval [0,1]
	return (double)rand()/((double)(((double)RAND_MAX)/((double)1.0)));
}
double gaussian(){
	double u1; double u2;

	u1 = uniformlyRandD();
	u2 = uniformlyRandD();
	double z1 = sqrt(-2 * log(u1))*cos(2*M_PI * u2);
	return z1;
}
