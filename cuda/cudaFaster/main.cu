#include "hip/hip_runtime.h"
#include "main.cuh"
#define Print 1
#define DontPrint 0
int main(){
	
	setlocale(LC_CTYPE, "");
	initMat();
	int m = 4091; int q = 4096; int n = 4091;
	matrix test1;
	matrix test2;
	matrix test3;
	matrix test4;
	matrixAllocate(&test1,m,q);
	matrixAllocate(&test2,q,n);
	matrixAllocate(&test3,m,n);
	matrixAllocate(&test4,m,n);
	matrixRandFill(&test2);
	matrixRandFill(&test1);
	clock_t start = clock();
	matrixMult(&test1,&test2,&test3);
	clock_t end = clock();
	printf("Time taken: %f seconds\n", ((double)(end - start)) / CLOCKS_PER_SEC);

	start = clock();
	matrixMultNoDelete(&test1,&test2,&test4);
	end = clock();
	printf("Time taken: %f seconds\n", ((double)(end - start)) / CLOCKS_PER_SEC);

	printf("Here lies dax \n");
	matrixEqual(&test3, &test4);
}
