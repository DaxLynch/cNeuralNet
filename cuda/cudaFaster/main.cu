#include "hip/hip_runtime.h"
#include "main.cuh"
#define Print 1
#define DontPrint 0
int main(){
	
	setlocale(LC_CTYPE, "");
	initMat();
	int size = 512;	
	matrix test1;
	matrix test2;
	matrix test3;
	matrix test4;
	matrixAllocate(&test1,size,size);
	matrixAllocate(&test2,size,size);
	matrixAllocate(&test3,size,size);
	matrixAllocate(&test4,size,size);
	matrixRandFill(&test2);
	matrixRandFill(&test1);
	clock_t start = clock();
	matrixMult(&test1,&test2,&test3);
	clock_t end = clock();
	printf("Time taken: %f seconds\n", ((double)(end - start)) / CLOCKS_PER_SEC);

	start = clock();
	matrixMultNoDelete(&test1,&test2,&test4);
	end = clock();
	printf("Time taken: %f seconds\n", ((double)(end - start)) / CLOCKS_PER_SEC);
	//matrixPrint(&test3);
	//printf("Here lies dax \n");
	//matrixPrint(&test4);

	printf("Here lies dax \n");
	matrixEqual(&test3, &test4);
}
