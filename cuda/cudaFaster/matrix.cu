#include "hip/hip_runtime.h"
void initMat(){
	srand(time(NULL));
}

void matrixAllocate(matrix* mat, int m, int n){
	
	mat->m = m;
	mat->n = n;
	hipMalloc(&mat->array, sizeof(float)*m*n);
	
}

int matrixCopy(matrix* dst, matrix* src){
	
	if ((dst->n != src->n) || (dst->m != src->m)){
		printf("(%d, %d) != (%d, %d)\n",dst->m,dst->n,src->m,src->n);
		printf("matrixCopy wrong dimension\n");
		return -1;
	}
	int m = dst->m; int n = dst->n;
	
	hipMemcpy(dst->array, src->array, m*n * sizeof(float), hipMemcpyDeviceToDevice);
	
	return 0;
}

void matrixRandFill(matrix* mat){ //Do not call if anything has been initilized ther budrick
	
	int m = mat->m; int n = mat->n;
	float* temp = (float*)malloc(sizeof(float)*m*n);
 	for(int i = 0; i < m; i++){
		for(int j = 0; j < n; j++){
			temp[i*n + j] = gaussian();
		}
	}
	hipMemcpy(mat->array, temp, sizeof(float)*m*n,hipMemcpyHostToDevice);
	free(temp);
	
	return;
}

void matrixScalar(matrix* mat, float scalar){
	
	int m = mat->m; int n = mat->n;
	dim3 threadsPerBlock(32,32);
	dim3 numBlocks(ceil(float(n) / 32.0f), ceil(float(m) / 32.0f));
	matscalar<<<numBlocks, threadsPerBlock>>>(mat->array, scalar, m, n);
	
}
__global__ void matscalar(float* A, float scalar, int m, int n){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if((x < n) && (y < m)){
		A[y*n + x] *= scalar;
	}
}

int matrixTranspose(matrix* mat){
	int m = mat->m; int n = mat->n;
	float* newArray = NULL;
	hipMalloc(&newArray, sizeof(float) * m*n);
	dim3 threadsPerBlock(32,32);
	dim3 numBlocks(ceil(float(n)/32.0f), ceil(float(m)/32.0f));

	mattrans<<<numBlocks, threadsPerBlock>>>(newArray, mat->array, m,n);
	mat->n = mat->m;
	mat->m = n;
	hipFree(mat->array);
	mat->array = newArray;
	
	return 0;
}
__global__ void mattrans(float* dst, float* src, int m, int n){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if((x < n) && (y < m)){
		dst[x*m + y] = src[y*n + x];
	}
}

int matrixSigmoid(matrix* A){
	
	if (A->n != 1){
		perror("Cannot do sigmoid");
		return -1;
	};
	dim3 threadsPerBlock(32,32);
	dim3 numBlocks(ceil(float(A->m)/32.0f), ceil(float(A->n)/32.0f));
	
	matsig<<<numBlocks, threadsPerBlock>>>(A->array, A->m,A->n);
	
	return 0;
}
__global__ void matsig(float* dst, int m, int n){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if((x < n) && (y < m)){
		dst[y*n + x] = 1.0f/(1.0f + exp(-dst[y*n + x]));
	}
}

int matrixSigmoidPrime(matrix* A){
	
	if (A->n != 1){
		perror("Cannot do sigmoid prime");
		return -1;
	};
	dim3 threadsPerBlock(32,32);
	dim3 numBlocks(ceil(float(A->m)/32.0f), ceil(float(A->n)/32.0f));

	matsigp<<<numBlocks, threadsPerBlock>>>(A->array, A->m,A->n);
	return 0;
}
__global__ void matsigp(float* dst, int m, int n){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if((x < n) && (y < m)){
		float temp = dst[y*n + x];
		dst[y*n + x] = 1.0f/(2.0f + exp(-temp) + exp(temp));
	}
}

int matrixMultTransFirst(matrix* A, matrix* B, matrix* out){
	if ((A->m != B->m)||(A->n != out->m)||(B->n != out->n)){
		printf("(%d, %d)T x (%d, %d) != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}
	int am = A->n;
       	int an = A->m;
       	int bn = B->n; 
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	
	matmulttrans1<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	
	return 0;
}

__global__ void matmulttrans1(float* A, float* B, float* C, int m, int q, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float temp = 0;
	if((x < n) && (y < m)){
		for(int i = 0; i < q; i++){
			temp += A[i*m + y] * B[i*n + x];	
		}
	C[y*n + x] = temp;
	}
}

int matrixMult(matrix* A, matrix* B, matrix* out){
	
	if ((A->n != B->m)||(A->m != out->m)||(B->n != out->n)){
		printf("(%d, %d) x (%d, %d) != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}
	
	int am = A->m;
       	int an = A->n;
       	int bn = B->n; 
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	
	matmult<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	
	return 0;
}

__global__ void matmult(float* A, float* B, float* C, int m, int q, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x ;
	int y = blockIdx.y * blockDim.y + threadIdx.y ;

	A += (blockIdx.y * 32  + threadIdx.y) * q  + threadIdx.x;
	B += (blockIdx.y * 32 + threadIdx.y) * n + (blockIdx.x * 32) + threadIdx.x ;
	C += (blockIdx.y * 32 * n) * (blockIdx.x * 32);
	
	float temp = 0;
		__shared__ float As[32*32];
		__shared__ float Bs[32*32];
		int i = 0;
		while( i < q - 31 ){
			As[threadIdx.y * 32 + threadIdx.x] = A[0];
			Bs[threadIdx.x * 32 + threadIdx.y] = B[0];
			__syncthreads();
			A+= 32;
			B += n;	

			for(int j = 0; j < 32; j++){
				temp += As[threadIdx.y * 32 + j] * Bs[threadIdx.x * 32 + j];	
			}
			i+= 32;
			__syncthreads();
		}

		if (i != q){
			As[threadIdx.y * 32 + threadIdx.x] = A[0];
			Bs[threadIdx.x * 32 + threadIdx.y] = B[0];
			__syncthreads();

			for(int j = 0; j < (q - i); j++){	
				temp += As[threadIdx.y * 32 + j] * Bs[ threadIdx.x * 32 + j];	
			}
		}				

			
		if((x < n) && (y < m)){
			C[y*n + x] = temp;	
		}
}

int matrixMultTransSecond(matrix* A, matrix* B, matrix* out){
	if ((A->n != B->n)||(A->m != out->m)||(B->m != out->n)){
		printf("(%d, %d) x (%d, %d)T != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}
	int am = A->m;
       	int an = A->n;
       	int bn = B->m; 
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	matmulttrans2<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	return 0;
}

__global__ void matmulttrans2(float* A, float* B, float* C, int m, int q, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float temp = 0;
	if((x < n) && (y < m)){
		for(int i = 0; i < q; i++){
			temp += A[y*q + i] * B[x*q + i];	
		}
	C[y*n + x] = temp;
	}
}

int matrixMultTransFirstNoDelete(matrix* A, matrix* B, matrix* out){
	if ((A->m != B->m)||(A->n != out->m)||(B->n != out->n)){
		printf("(%d, %d)T x (%d, %d) != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}
	int am = A->n;
       	int an = A->m;
       	int bn = B->n; 

	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	
	matmulttrans1D<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	
	return 0;
}

__global__ void matmulttrans1D(float* A, float* B, float* C, int m, int q, int n){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float temp = 0;
	if((x < n) && (y < m)){
		for(int i = 0; i < q; i++){
			temp += A[i*m + y] * B[i*n + x];	
		}
	C[y*n + x] += temp;
	}
}

int matrixMultNoDelete(matrix* A, matrix* B, matrix* out){
	
	if ((A->n != B->m)||(A->m != out->m)||(B->n != out->n)){
		printf("(%d, %d) x (%d, %d) != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}
	
	int am = A->m;
       	int an = A->n;
       	int bn = B->n; 
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	
	matmultD<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	
	return 0;
}

__global__ void matmultD(float* A, float* B, float* C, int m, int q, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float temp = 0;
	if((x < n) && (y < m)){
		for(int i = 0; i < q; i++){
			temp += A[y*q + i] * B[i*n + x];	
		}
	C[y*n + x] = temp;
	
	}
}

int matrixMultTransSecondNoDelete(matrix* A, matrix* B, matrix* out){
	if ((A->n != B->n)||(A->m != out->m)||(B->m != out->n)){
		printf("(%d, %d) x (%d, %d)T != (%d, %d)",A->m,A->n,B->n,B->m,out->m,out->n);
		perror("matrixMult error: sizes incorrect");
		return -1;
	}

	int am = A->m;
       	int an = A->n;
       	int bn = B->m; 
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(bn) / 32.0f), ceil(float(am) / 32.0f));
	
	matmulttrans2D<<<numBlocks, threadsPerBlock>>>(A->array,B->array,out->array,am,an,bn);
	
	return 0;
}

__global__ void matmulttrans2D(float* A, float* B, float* C, int m, int q, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float temp = 0;
	if((x < n) && (y < m)){
		for(int i = 0; i < q; i++){
			temp += A[y*q + i] * B[x*q + i];	
		}
	C[y*n + x] += temp;
	}
}


int matrixAdd(matrix* dst, matrix* src){ //in place,a s opposed to matrix multiply???
	
	if ((dst->n != src->n) || (dst->m != src->m)){
		perror("matrixAdd wrong dimension");
		return -1;
	}
	int m = dst->m; int n = dst->n;
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(n) / 32.0f), ceil(float(m) / 32.0f));

	matadd<<<numBlocks, threadsPerBlock>>>(dst->array,src->array,m,n);
	
	return 0;
}

__global__ void matadd(float* A, float* B, int m, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if((x < n) && (y < m)){
		A[y*n + x] += B[y*n + x];	
	}
}

int matrixHamProd(matrix* dst, matrix* src){ //in place,a s opposed to matrix multiply???
	
	if ((dst->n != src->n) || (dst->m != src->m)){
		perror("matrixAdd wrong dimension");
		return -1;
	}
	int m = dst->m; int n = dst->n;
	
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(ceil(float(n) / 32.0f), ceil(float(m) / 32.0f));

	matham<<<numBlocks, threadsPerBlock>>>(dst->array,src->array,m,n);
	
	return 0;
}

__global__ void matham(float* A, float* B, int m, int n){ //basic implementation
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if((x < n) && (y < m)){
		A[y*n + x] *= B[y*n +x];	
	}
}

void matrixPrint(matrix* mat) {
	hipDeviceSynchronize();	
	int m = mat->m; int n = mat->n;
	float* temp = (float*)malloc(sizeof(float)*m*n);
	
	hipMemcpy(temp, mat->array, sizeof(float)*m*n,hipMemcpyDeviceToHost);
	printf("m: %d n: %d \n", m,n);
    	for (int i = 0; i < m; i++) {
        	for (int j = 0; j < n; j++) {
        	  	  printf(" %f ", temp[i*n + j]);
        	}
        	printf("\n");
    	}
	free(temp);
	
}



int matrixNonZeros(matrix* mat) {
	hipDeviceSynchronize();	
	int m = mat->m; int n = mat->n;
	float* temp = (float*)malloc(sizeof(float)*m*n);
	
	hipMemcpy(temp, mat->array, sizeof(float)*m*n,hipMemcpyDeviceToHost);
	int nonzeros = 0;
    	for (int i = 0; i < m; i++) {
        	for (int j = 0; j < n; j++) {
        		if(temp[i*n + j]){
				nonzeros++;
			}	
		}
    	}
	free(temp);
	printf("%d non zero \n", nonzeros);
	return nonzeros;
}
int matrixEqual(matrix* A, matrix* B) {
	hipDeviceSynchronize();	
	int m = A->m; int n = A->n;
	float* temp1 = (float*)malloc(sizeof(float)*m*n);
	float* temp2 = (float*)malloc(sizeof(float)*m*n);
	
	hipMemcpy(temp1, A->array, sizeof(float)*m*n,hipMemcpyDeviceToHost);
	hipMemcpy(temp2, B->array, sizeof(float)*m*n,hipMemcpyDeviceToHost);
	int nonequals = 0;
    	for (int i = 0; i < m; i++) {
        	for (int j = 0; j < n; j++) {
        		if(temp1[i*n + j] - temp2[i*n + j]  >  .01f ){
				printf("i: %d, j: %d, ,%f != %f \n", i,j,temp1[i * n + j], temp2[i * n + j]);
				nonequals++;
			}	
		}
    	}
	free(temp1);
	free(temp2);
	printf("%d non equal \n", nonequals);
	return nonequals;
}
void matrixFree(matrix* mat) {
	
    hipFree(mat->array); // Free the array of rows
	
}


