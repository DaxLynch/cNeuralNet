#include "hip/hip_runtime.h"
void networkAllocate(network* net, int len, ...){
	net->num_layers = len;
	net->sizes = (int*)malloc(sizeof(int)*len);
	net->weights = (matrix*)malloc(sizeof(matrix)*(len-1));
	net->biases = (matrix*)malloc(sizeof(matrix)*(len-1));

	va_list args;
	va_start(args, len);
	for(int i = 0; i < len; i++){
		net->sizes[i] = va_arg(args, int);
	}
	va_end(args);
	
	for(int i = 0; i < len - 1; i++){
		matrixAllocate(&net->weights[i], net->sizes[i+1], net->sizes[i]);
		matrixAllocate(&net->biases[i], net->sizes[i+1], 1);
	}
}

void networkWeightsInit(network* net){
	for(int i = 0; i < net->num_layers - 1; i++){
		matrixRandFill(&net->weights[i]);
		matrixRandFill(&net->biases[i]);
	}
	//matrixPrint(&net->weights[0]);
}

void networkPrint(network* net){
	for(int i = 0; i < net->num_layers - 1; i++){
		printf("net->weights[%d]\n", i);
		matrixPrint(&net->weights[i]);
	}

}



void networkSizeAllocate(network* net, network* src){
	net->num_layers = src->num_layers;
	int len = net->num_layers;
	net->sizes = (int*)malloc(sizeof(int)*len);
	net->weights = (matrix*)malloc(sizeof(matrix)*(len-1));
	net->biases = (matrix*)malloc(sizeof(matrix)*(len-1));

	for(int i = 0; i < len; i++){
		net->sizes[i] = src->sizes[i];
	}
	
	for(int i = 0; i < len - 1; i++){
		matrixAllocate(&net->weights[i], src->weights[i].m, src->weights[i].n);
		matrixAllocate(&net->biases[i], src->biases[i].m, 1);
	}	
}

void networkFree(network* net){	
	int len = net->num_layers;
	for(int i = 0; i < len - 1; i++){
		matrixFree(&net->weights[i]);
		matrixFree(&net->biases[i]);
	}
	free(net->sizes);
	free(net->weights);
	free(net->biases);
}




void update_mini_batch(network* net, int batch_size, int* batch_list, data* data_set, double eta){
	network nabla;
	int len = net->num_layers;
	networkSizeAllocate(&nabla, net);
	for(int i = 0; i < batch_size; i++){
		backprop(&nabla, net, batch_list[i], data_set);
	}
	for(int i = 0; i < len - 1; i++){
		matrixScalar(&nabla.weights[i], -eta/batch_size);
		matrixScalar(&nabla.biases[i], -eta/batch_size);
		matrixAdd(&net->weights[i],&nabla.weights[i]);
		matrixAdd(&net->biases[i],&nabla.biases[i]);
	}
	networkFree(&nabla);
}

void shuffle(int *array, size_t n)
{
    if (n > 1) 
    {
        size_t i;
        for (i = 0; i < n - 1; i++) 
        {
          size_t j = i + rand() / (RAND_MAX / (n - i) + 1);
          int t = array[j];
          array[j] = array[i];
          array[i] = t;
        }
    }
}

void networkSGD(network* net, data* dataSet, int dataLength, data* testSet, int testLength, int print, int epochs, int batch_size, double eta){
	for (int j = 0; j < epochs; j++){
		if (print) {printf("Starting epoch:%d \n", j);};
		clock_t start = clock();
		int* shuffled = (int*)malloc(sizeof(int) * dataLength);
		for(int i = 0; i < dataLength; i++){
			shuffled[i] = i;
		}
		
		shuffle(shuffled, dataLength);
		for( int i = 0; i < dataLength/batch_size; i++){
			update_mini_batch(net, batch_size,  shuffled + (batch_size*i), dataSet, eta);
		
		}
		clock_t end = clock();
		if (print){
			printf("Time taken: %f seconds\n", ((double)(end - start)) / CLOCKS_PER_SEC);
			evaluateSet(net, testSet, testLength, 1);
		}
		free(shuffled);
	}	
}

