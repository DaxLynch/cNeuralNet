void displayChar(unsigned char input){	
	char ret[] = {' ', '-','*','#','%','@'};
	printf("%c", ret[input/43]);
}

void dataLoader(data** dataPointer, char* images, char* labels, int dataLength){
	FILE* training_images = fopen(images, "rb");
	FILE* training_labels = fopen(labels, "rb");
	if(training_images == NULL){
		printf("fuck");
		exit(EXIT_FAILURE);
	}
	unsigned char bs[16];
	*dataPointer = (data*)malloc(sizeof(data) * dataLength);
	fread(bs, 16, 1, training_images);
	fread(bs, 8, 1, training_labels);
	unsigned char buff[28*28];
	float buff2[28*28];
	for(int m = 0; m < dataLength; m++){
		matrixAllocate(&((*dataPointer)[m].matrix), 784, 1);	
		fread(buff, 1, 28*28, training_images);
		for(int i = 0; i < (28 * 28); i ++){
			buff2[i] = (((float)buff[i] - 33.0f)/255.0f);
		}
		hipMemcpy((*dataPointer)[m].matrix.array, buff2, 28*28*sizeof(float), hipMemcpyHostToDevice);
		unsigned char temp;
		fread(&temp, sizeof(unsigned char), 1, training_labels);
		(*dataPointer)[m].truth = (int)temp;
	}
	fclose(training_images);
	fclose(training_labels);
}
void fileDataViewer(char* inputFile){ //"trainingData/train-images.idx3-ubyte",
	FILE* training_images = fopen(inputFile, "rb");
	if (training_images == NULL){
		printf("stupid bs\n");
		return;
	}
	unsigned char bs[16];
	fread(bs, 16, 1, training_images);
	unsigned char array[28 * 28];	
	while(getchar()){
		fread(array, 28*28, 1, training_images);
		for(int i = 0; i < 28; i++){
			for(int j = 0; j < 28; j++){
				displayChar(array[i*28 + j]);
			}
			printf("\n");
		}
	}
	printf("\n");
	fclose(training_images);
}

void structDataViewer(data* dataPointer){
	float temp[28*28];
	hipMemcpy(temp, dataPointer->matrix.array, 28*28*sizeof(float), hipMemcpyDeviceToHost);
	for(int i = 0; i < 28; i++){
		for(int j = 0; j < 28; j++){
			displayChar((unsigned char)(temp[i*28 + j]*255.0f  + 33.0f));
		}
		printf("\n");
	}
	printf("\n");
}


void dataFree(data** dataPointer, int dataLength){
	for(int m = 0; m < dataLength; m++){
		matrixFree(&((*dataPointer)[m].matrix));
	}
	free(*dataPointer);
}
