#include "hip/hip_runtime.h"
#include "dataLoader.cuh"
using namespace std;
data::data(const std::string &name, int length): dataMatrix(784, length){
	dataLength = length;

	ifstream file;
	file.open(name);	
	file.seekg(16);
	
	char *charBuff = new char[length*784];
	file.read(charBuff, length*784);
	unsigned char *cudaCharBuff;
	hipMalloc(&cudaCharBuff,sizeof(char) * length * 784);
	hipMemcpy(cudaCharBuff, charBuff, length * 784, hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(1024);
	dim3 blocks(ceil((float)length/1024.0f));
	castIntToFloat<<<blocks,threadsPerBlock>>>(dataMatrix.array, cudaCharBuff, length);
	hipDeviceSynchronize();	
	hipFree(cudaCharBuff);
	delete[] charBuff;
}


ostream& operator<<(ostream& os, const data& dat){
	return os << dat.dataMatrix;
}

void data::displayChar(unsigned char input){
	char ret[] = {' ', '-', '*', '#', '%', '@'};
	cout << ret[input/43];
}

void data::print(int entry){
	float temp[28*28];
	hipMemcpy(temp, dataMatrix.array + entry*784, 28*28*sizeof(float), hipMemcpyDeviceToHost);
	for(int i = 0; i < 28; i++){
		for(int j = 0; j < 28; j++){
			displayChar((unsigned char)(temp[i*28 + j]*255.0f  + 33.0f));
		}
		cout << endl;
	}
	cout << endl;
}


__global__ void castIntToFloat(float *dst, unsigned char *src, int length){
	int x = threadIdx.x + blockIdx.x * 1024.0f;
	if ( x < length){
		for(int i = 0; i < 28*28; i++){
			dst[x*784 + i] = ((float)src[x*784 + i] - 33.0f)/255.0f;
		}
	}
}
