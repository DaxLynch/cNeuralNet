#include "matrix.cuh"
using namespace std;
matrix::matrix(int rows, int cols){
	m = rows;
	n = cols;
	hipMalloc(&array, m*n*sizeof(float));
};
matrix::~matrix(){
	hipFree(array);
}
ostream& operator<<(ostream& os, matrix const &mat){
	int m = mat.m; int n = mat.n;
	float* temp = new float[m*n];
	hipMemcpy(temp, mat.array, m*n*sizeof(float), hipMemcpyDeviceToHost);
	for(int i = 0; i < m; i++){
		for(int j = 0; j < n; j++){
			os << temp[   j*m   +   i] << ' ';
		}
		os << '\n';
	}
	return os;
}




