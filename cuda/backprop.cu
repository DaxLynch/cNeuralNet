#include "hip/hip_runtime.h"
__global__ void matsubone(float* array, int y){
	array[y] -= 1;
}

void backprop(network* nabla, network* net, int x, data* batch_list){
	int len = net->num_layers;
	static int allocated = 1;
	static matrix *activations;
        static matrix *zactivations;
	static matrix *z;
	static matrix *delta;

	if (allocated){
		activations = (matrix*)malloc(sizeof(matrix)*len); 
		zactivations = (matrix*)malloc(sizeof(matrix)*(len - 1)); 
		z = (matrix*)malloc(sizeof(matrix)*(len - 1)); 
		delta = (matrix*)malloc(sizeof(matrix)*(len - 1)); 
		matrixAllocate(&activations[0], net->sizes[0],1);
		for (int i = 1; i < len; i++){
			matrixAllocate(&zactivations[i-1], net->sizes[i], 1);
			matrixAllocate(&z[i-1], net->sizes[i], 1);
			matrixAllocate(&delta[i-1], net->sizes[i], 1);
			matrixAllocate(&activations[i], net->sizes[i], 1); 
		}

		allocated = 0;
	}
	matrixCopy(&activations[0], &batch_list[x].matrix);


	for(int i = 1; i < len; i++){

		matrixMultAndDelete(&net->weights[i-1], &activations[i-1], &z[i-1]);
		
		matrixAdd(&z[i-1], &net->biases[i-1]);

		matrixCopy(&zactivations[i-1], &z[i-1]);

		matrixSigmoid(&z[i-1]);
		matrixCopy(&activations[i], &z[i-1]);
	}
	int y = batch_list[x].truth; //only relevant to this data SHould be changed to a truth vector

	//wprintf(L"Y: %d",y);	

	//matrixAllocate(&delta, net->sizes[len-1], 1);
//	matrixPrint(&delta);
	matrixCopy(&delta[1], &activations[len-1]);

	matsubone<<<1,1>>>(delta[1].array, y);

	matrixSigmoidPrime(&zactivations[len-2]);
	matrixHamProd(&delta[1], &zactivations[len-2]);

	matrixAdd(&nabla->biases[len-2], &delta[1]); //add size check for mat copy
	
	matrixMultTransSecondAndDelete(&delta[1], &activations[len-2], &nabla->weights[len-2]);

	for(int i = len - 2; i > 0; i--){
	//	matrix weightsT;
	
	//	matrixAllocate(&weightsT, net->weights[i].m, net->weights[i].n); //free me
	//	matrixCopy(&weightsT, &net->weights[i]);
	//	matrixTranspose(&weightsT);

		matrixSigmoid(&zactivations[i-1]);


		matrixMultTransFirstAndDelete(&net->weights[i],&delta[i], &delta[i-1]);
		
		matrixHamProd(&delta[i-1], &zactivations[i-1]);

		matrixAdd(&nabla->biases[i-1], &delta[i-1]);
		matrixMultTransSecondAndDelete(&delta[i-1], &activations[i-1], &nabla->weights[i-1]);
	}


	return;
}
