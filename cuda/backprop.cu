#include "hip/hip_runtime.h"
__global__ void matsubone(float* array, int y){
	array[y] -= 1;
}

void backprop(network* nabla, network* net, int x, data* batch_list){
	int len = net->num_layers;

	static int allocated = 1;
	static matrix *activations;
        static matrix *zactivations;
	static matrix *delta;

	if (allocated){
		activations = (matrix*)malloc(sizeof(matrix)*len); 
		zactivations = (matrix*)malloc(sizeof(matrix)*(len - 1)); 
		delta = (matrix*)malloc(sizeof(matrix)*(len - 1)); 
		matrixAllocate(&activations[0], net->sizes[0],1);
		for (int i = 1; i < len; i++){
			matrixAllocate(&zactivations[i-1], net->sizes[i], 1);
			matrixAllocate(&delta[i-1], net->sizes[i], 1);
			matrixAllocate(&activations[i], net->sizes[i], 1); 
		}
		allocated = 0;
	}
	matrixCopy(&activations[0], &batch_list[x].matrix); 

	for(int i = 1; i < len; i++){
		matrixMult(&net->weights[i-1], &activations[i-1], &zactivations[i-1]);	
		matrixAdd(&zactivations[i-1], &net->biases[i-1]);
		matrixCopy(&activations[i], &zactivations[i-1]);
		matrixSigmoid(&activations[i]);
	}
	int y = batch_list[x].truth; //only relevant to this data SHould be changed to a truth vector
	matrixCopy(&delta[len -2], &activations[len-1]);

	matsubone<<<1,1>>>(delta[len -2].array, y); //these ops can be combined

	matrixSigmoidPrime(&zactivations[len-2]);
	matrixHamProd(&delta[len - 2], &zactivations[len-2]);
	matrixAdd(&nabla->biases[len-2], &delta[len -2]); //add size check for mat copy
	
	matrixMultTransSecondNoDelete(&delta[len -2], &activations[len-2], &nabla->weights[len-2]);

	for(int i = len - 2; i > 0; i--){

		matrixSigmoidPrime(&zactivations[i-1]);
		matrixMultTransFirst(&net->weights[i],&delta[i], &delta[i-1]);
		
		matrixHamProd(&delta[i-1], &zactivations[i-1]);

		matrixAdd(&nabla->biases[i-1], &delta[i-1]);
		matrixMultTransSecondNoDelete(&delta[i-1], &activations[i-1], &nabla->weights[i-1]);
	}
	//for(int i = 0; i < 2; i++){
	//	hipMemset(zactivations[i].array, 0, net->sizes[i+1] * sizeof(float)* 1);
	//	hipMemset(activations[i].array, 0, net->sizes[i] * sizeof(float)* 1);
	//	hipMemset(delta[i].array, 0, net->sizes[i+1] * sizeof(float)* 1);
	//
	//hipMemset(activations[2].array, 0, net->sizes[2] * sizeof(float)* 1);
	hipDeviceSynchronize();
	return;
}
