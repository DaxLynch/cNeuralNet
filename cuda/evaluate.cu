#include "hip/hip_runtime.h"
int evaluate(network* net, data* datum, int print){
	int len = net->num_layers;
	matrix* activations = (matrix*)malloc(sizeof(matrix)*len); 
	matrixAllocate(&activations[0], net->sizes[0],1);
	matrixCopy(&activations[0], &datum->matrix);
	
	for(int i = 1; i < len; i++){
		matrixAllocate(&activations[i], net->sizes[i], 1);
		matrixMult(&net->weights[i-1], &activations[i-1], &activations[i]);
		matrixAdd(&activations[i], &net->biases[i-1]);

		matrixSigmoid(&activations[i]);
	}
	double max = 0;
	int maxArg = 0;
	float temp[10]; 
	hipMemcpy(temp, activations[len -1].array, 10 *sizeof(float), hipMemcpyDeviceToHost);
	for(int i = 0; i < 10; i++){
		if (temp[i] > max){
			max = temp[i];
			maxArg = i;
		}
	}
	if(print){
		structDataViewer(datum);
		printf("returned %d with value of %.2lf, true value is %d\n", maxArg, max, datum->truth);
	}
	for(int i = 0; i < len; i++){
		matrixFree(&activations[i]);
	}
	free(activations);
	return maxArg;
}
int evaluateSet(network* net, data* testData, int dataLength){
	int correct = 0;
	for(int i = 0; i < dataLength; i++){
		if (evaluate(net, &testData[i], 0)  == testData[i].truth){
			correct++;
		}
	}
	printf("%.2f %% correct, %d/%d \n", ((double)correct)/((double)dataLength)*100.0, correct, dataLength);
	return 0;
}
int evaluateSetManual(network* net, data* datum, int dataLength){
	printf("Press q to exit, press any character to see the next evaluation\n");
	int i = 0;
	while(getchar() != 'q' || i == dataLength ){ 
		evaluate(net, datum + i, 1);
		i++;
	}
	return 0;
}
int evaluateSetFailures(network* net, data* datum, int dataLength){
	printf("Press q to exit, press any character to see the next failed evaluation\n");
	int i = 0;
	while(getchar() != 'q' || i == dataLength ){ 
		for(;evaluate(net, datum+i,0)==datum[i].truth  && i < 60000; i++){
		}
		evaluate(net, datum + i,1);
		i++;
	}
	return 0;
}


